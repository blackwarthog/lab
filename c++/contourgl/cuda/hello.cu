
#include <hip/hip_runtime.h>
extern "C" {

__constant__ char s[] = "Hello!";


__global__ void hello(char *out) {
	int i = threadIdx.x;
	out[i] = s[i];
}

}