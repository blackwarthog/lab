
#include <hip/hip_runtime.h>
/*
    ......... 2018 Ivan Mahonin

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

//extern "C" {

#define ONE       65536
#define TWO      131072               // (ONE)*2
#define HALF      32768               // (ONE)/2
#define ONE_F     65536.f             // (float)(ONE)
#define DIV_ONE_F 0.0000152587890625f // 1.f/(ONE_F)


__global__ void path(
	int width,
	int height,
	int *marks,
	const float2 *points,
	int begin,
	int end,
	int minx )
{
	int id = blockIdx.x*blockDim.x + threadIdx.x + begin;
	if (id >= end) return;
	float2 p0 = points[id];
	float2 p1 = points[id + 1];
	
	bool flipx = p1.x < p0.x;
	bool flipy = p1.y < p0.y;
	if (flipx) { p0.x = (float)width  - p0.x; p1.x = (float)width  - p1.x; }
	if (flipy) { p0.y = (float)height - p0.y; p1.y = (float)height - p1.y; }
	float2 d;
	d.x = p1.x - p0.x;
	d.y = p1.y - p0.y;
	int w1 = width - 1;
	int h1 = height - 1;
	float kx = d.x/d.y;
	float ky = d.y/d.x;
	
	while(p0.x != p1.x || p0.y != p1.y) {
		int ix = max((int)p0.x, 0);
		int iy = (int)p0.y;
		if (ix > w1) return;

		float2 px, py;
		px.x = (float)(ix + 1);
		py.y = (float)(iy + 1);
		iy = max(0, min(h1, iy));
		
		px.y = p0.y + ky*(px.x - p0.x);
		py.x = p0.x + kx*(py.y - p0.y);
		
		float2 pp1 = p1;
		if (pp1.x > px.x) pp1 = px;
		if (pp1.y > py.y) pp1 = py;
		
		float cover = (pp1.x - p0.x)*ONE_F;
		float area = py.y - 0.5f*(p0.y + pp1.y);
		if (flipx) { ix = w1 - ix; cover = -cover; }
		if (flipy) { iy = h1 - iy; area = 1.f - area; }
		p0 = pp1;
		
		atomicAdd(
			(unsigned long long*)(marks + 2*(iy*width + ix)),
			((unsigned long long)(unsigned int)(int)(cover) << 32)
			| (unsigned long long)(unsigned int)((int)(area*cover)) );
		//int *mark = marks + ((iy*width + ix) << 1);
		//atomicAdd(mark, (int)(area*cover));
		//atomicAdd(mark + 1, (int)(cover));
	}
}

__global__ void fill(
	int width,
	int2 *marks,
	float4 *image,
	float4 color,
	int4 bounds )
{
	int id = blockIdx.x*blockDim.x + threadIdx.x + bounds.x;
	if (id >= bounds.z) return;
	id += bounds.y*width;
	marks += id;
	image += id;

	int icover = 0;
	while(true) {
		int2 m = *marks;
		*marks = make_int2(0, 0);
		float alpha = (float)abs(m.x + icover)*color.w*DIV_ONE_F;
		marks += width;

		icover += m.y;
		float one_alpha = 1.f - alpha;
		
		float4 p = *image;
		p.x = p.x*one_alpha + color.x*alpha;
		p.y = p.y*one_alpha + color.y*alpha;
		p.z = p.z*one_alpha + color.z*alpha;
		p.w = p.w*one_alpha + color.w*alpha;
		*image = p;
		
		if (++bounds.y >= bounds.w) return;
		image += width;
	}
}

//}