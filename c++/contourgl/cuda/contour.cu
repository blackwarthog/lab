
#include <hip/hip_runtime.h>
/*
    ......... 2018 Ivan Mahonin

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

extern "C" {

#define ONE       65536
#define TWO      131072               // (ONE)*2
#define HALF      32768               // (ONE)/2
#define ONE_F     65536.f             // (float)(ONE)
#define DIV_ONE_F 0.0000152587890625f // 1.f/(ONE_F)


__global__ void clear(
	int width,
	int height,
	int4 *marks )
{
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	int c = id % width;
	marks[id] = make_int4(0, 0, c | (c + 1), 0);
}

__global__ void path(
	int width,
	int height,
	int *marks,
	const float2 *points,
	int begin,
	int end,
	int minx )
{
	int id = blockIdx.x*blockDim.x + threadIdx.x + begin;
	if (id >= end) return;
	float2 p0 = points[id];
	float2 p1 = points[id + 1];
	
	bool flipx = p1.x < p0.x;
	bool flipy = p1.y < p0.y;
	if (flipx) { p0.x = (float)width - p0.x; p1.x = (float)width - p1.x; }
	if (flipy) { p0.y = (float)height - p0.y; p1.y = (float)height - p1.y; }
	
	float2 d;
	d.x = p1.x - p0.x;
	d.y = p1.y - p0.y;
	float kx = d.x/d.y;
	float ky = d.y/d.x;
	int w1 = width - 1;
	int h1 = height - 1;
	
	while(p0.x != p1.x || p0.y != p1.y) {
		int ix = (int)p0.x;
		int iy = max((int)p0.y, 0);
		if (iy > h1) return;

		float2 px, py;
		px.x = (float)(ix + 1);
		py.y = (float)(iy + 1);
		ix = max(0, min(w1, ix));
		
		px.y = p0.y + ky*(px.x - p0.x);
		py.x = p0.x + kx*(py.y - p0.y);

		float2 pp1 = p1;
		if (pp1.x > px.x) pp1 = px;
		if (pp1.y > py.y) pp1 = py;
		
		float cover = (pp1.y - p0.y)*ONE_F;
		float area = px.x - 0.5f*(p0.x + pp1.x);
		if (flipx) { ix = w1 - ix; area = 1.f - area; }
		if (flipy) { iy = h1 - iy; cover = -cover; }
		
		int *row = marks + 4*iy*width;
		atomicAdd(
			(unsigned long long*)(row + 4*ix),
			((unsigned long long)(unsigned int)(int)(cover) << 32)
			| (unsigned long long)(unsigned int)((int)(area*cover)) );
		//row[4*ix] += (int)(area*cover);
		//row[4*ix + 1] += (int)(cover);
		//atomicAdd(row + 4*ix, (int)(area*cover));
		//atomicAdd(row + 4*ix + 1, (int)(cover));
		
		row += 2;
		int iix = (ix & (ix + 1)) - 1;
		while(iix >= minx) {
			atomicMin(row + 4*iix, ix);
			iix = (iix & (iix + 1)) - 1;
		}
		
		p0 = pp1;
	}
}

__global__ void fill(
	int width,
	int4 *marks,
	float4 *image,
	float4 color,
	int4 bounds )
{
	int id = blockIdx.x*blockDim.x + threadIdx.x + bounds.y;
	if (id >= bounds.w) return;
	id *= width;
	marks += id;
	image += id;
	
	int4 *mark;
	float4 *pixel;

	int4 m;
	int icover = 0, c0 = bounds.x, c1 = bounds.x;
	while(c1 < bounds.z) {
		if (abs(icover) > HALF)
			while(c0 < c1)
				image[c0++] = color;

		mark = &marks[c1];
		m = *mark;
		*mark = make_int4(0, 0, c1 | (c1 + 1), 0); 
		
		float alpha = (float)abs(m.x + icover)*DIV_ONE_F;
		float one_alpha = 1.f - alpha;
		
		pixel = &image[c1];
		float4 p = *pixel;
		p.x = p.x*one_alpha + color.x*alpha;
		p.y = p.y*one_alpha + color.y*alpha;
		p.z = p.z*one_alpha + color.z*alpha;
		p.w = p.w*one_alpha + color.w*alpha;
		*pixel = p;
		
		icover += m.y;
		c0 = c1 + 1;
		c1 = m.z;
	}
}

}